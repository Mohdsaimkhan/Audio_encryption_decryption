#include "hip/hip_runtime.h"
#include <iostream>
#include <sndfile.hh>
#include <cmath>
#include <hip/hip_runtime.h>

// Function to design a digital low-pass filter (Gaussian FIR)
void designLowPassFilter(float* filterCoefficients, int filterLength, float cutoffFrequency) {
    const float sigma = 0.1; 
    const float twoSigmaSquare = 2.0f * sigma * sigma;
    const int midPoint = filterLength / 2;

    for (int i = 0; i < filterLength; ++i) {
        int distance = i - midPoint;
        filterCoefficients[i] = std::exp(-(distance * distance) / twoSigmaSquare);
    }

    // Normalize the filter coefficients
    float sum = 0.0f;
    for (int i = 0; i < filterLength; ++i) {
        sum += filterCoefficients[i];
    }

    for (int i = 0; i < filterLength; ++i) {
        filterCoefficients[i] /= sum;
    }
}

// CUDA kernel for applying the low-pass filter
__global__ void applyLowPassFilter(float* data, int dataSize, const float* filterCoefficients, int filterLength) {
    __shared__ float sharedData[256]; // Shared memory for input data block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dataSize) {
        sharedData[threadIdx.x] = data[idx];
        __syncthreads();

        float result = 0.0f;

        for (int i = 0; i < filterLength; ++i) {
            int dataIndex = threadIdx.x - i + filterLength / 2;
            if (dataIndex >= 0 && dataIndex < blockDim.x) {
                result += sharedData[dataIndex] * filterCoefficients[i];
            }
        }

        data[idx] = result;
    }
}

// CUDA kernel for modulation (multiplication with sinusoid)
__global__ void applyModulation(float* data, int dataSize, float modulationFrequency, float sampleRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dataSize) {
        float angle = 2.0f * M_PI * modulationFrequency * idx / sampleRate;
        data[idx] *= sin(angle);
    }
}

// Function to perform audio encryption on GPU and measure time
float audioEncryptionGPU(float* d_data, int dataSize, float cutoffFrequency, float modulationFrequency, float sampleRate) {
    const int filterLength = 64; 

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host variable for filter coefficients
    float h_filterCoefficients[filterLength];
    
    // Design the low-pass filter on the host
    designLowPassFilter(h_filterCoefficients, filterLength, cutoffFrequency);

    // Allocate GPU memory for filter coefficients
    float* d_filterCoefficients;
    hipMalloc((void**)&d_filterCoefficients, filterLength * sizeof(float));

    // Copy filter coefficients to GPU
    hipMemcpy(d_filterCoefficients, h_filterCoefficients, filterLength * sizeof(float), hipMemcpyHostToDevice);

    // Configure GPU execution parameters
    const int blockSize = 256;
    const int gridSize = (dataSize + blockSize - 1) / blockSize;

    // Record start time
    hipEventRecord(start);

    // Launch the low-pass filter kernel on GPU
    applyLowPassFilter<<<gridSize, blockSize>>>(d_data, dataSize, d_filterCoefficients, filterLength);
    hipDeviceSynchronize();

    // Launch the modulation kernel on GPU
    applyModulation<<<gridSize, blockSize>>>(d_data, dataSize, modulationFrequency, sampleRate);
    hipDeviceSynchronize();

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and return elapsed time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Free GPU memory for filter coefficients
    hipFree(d_filterCoefficients);

    return milliseconds;
}


int main(int argc, char* argv[]) {
    // Parse command-line options to determine encryption or decryption
    bool encryptionMode = true; // Set to true for encryption, false for decryption

    // Check command-line arguments
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <input_file.wav> <output_file.wav> <cutoff_frequency> <modulation_frequency>" << std::endl;
        return 1;
    }

    // Load the audio file using libsndfile
    SndfileHandle inputFile(argv[1], SFM_READ);
    if (!inputFile) {
        std::cerr << "Error: Failed to open input file." << std::endl;
        return 1;
    }

    // Get audio file parameters
    int dataSize = static_cast<int>(inputFile.frames());
    int sampleRate = inputFile.samplerate();
    int numChannels = inputFile.channels();

    // Allocate host memory for audio data
    float* h_audioData = new float[dataSize];

    // Read audio data from the file
    inputFile.read(h_audioData, dataSize);

    // Allocate device memory for audio data
    float* d_audioData;
    hipMalloc((void**)&d_audioData, dataSize * sizeof(float));
    hipMemcpy(d_audioData, h_audioData, dataSize * sizeof(float), hipMemcpyHostToDevice);

    // Apply audio encryption on GPU and measure time
    float cutoffFrequency = std::stof(argv[3]); // Cutoff frequency from command line
    float modulationFrequency = std::stof(argv[4]); // Modulation frequency from command line
    float processingTime = audioEncryptionGPU(d_audioData, dataSize, cutoffFrequency, modulationFrequency, sampleRate);

    // Print processing time
    std::cout << "GPU Processing Time: " << processingTime << " ms" << std::endl;

    // Copy results from device to host
    hipMemcpy(h_audioData, d_audioData, dataSize * sizeof(float), hipMemcpyDeviceToHost);

    // Write the resulting WAV file using libsndfile
    SndfileHandle outputFile(argv[2], SFM_WRITE, SF_FORMAT_WAV | SF_FORMAT_PCM_16, numChannels, sampleRate);
    if (!outputFile) {
        std::cerr << "Error: Failed to open output file." << std::endl;
        return 1;
    }
    outputFile.write(h_audioData, dataSize);

    // Deallocate device memory
    hipFree(d_audioData);

    // Clean up
    delete[] h_audioData;

    return 0;
}
